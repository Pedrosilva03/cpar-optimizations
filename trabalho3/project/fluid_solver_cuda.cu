#include "hip/hip_runtime.h"
#include "fluid_solver.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>
#include <iostream> // For debugging output

#define IX(i, j, k) ((i) + (M + 2) * (j) + (M + 2) * (N + 2) * (k))
#define SWAP(x0, x)                                                            \
  {                                                                            \
    float *tmp = x0;                                                           \
    x0 = x;                                                                    \
    x = tmp;                                                                   \
  }
#define MAX(a, b) (((a) > (b)) ? (a) : (b))

int compute_size(int M, int N, int O) {
    return (M + 2) * (N + 2) * (O + 2);
}

static int global_size;

// Fluid simulation arrays
float *d_u, *d_v, *d_w, *d_u_prev, *d_v_prev, *d_w_prev;
float *d_dens, *d_dens_prev;

// Mallocs constantes dos kernels
void initCudaMalloc(int M, int N, int O){
    global_size = compute_size(M, N, O);
    int size = global_size * sizeof(float);

    if (hipMallocManaged((void**)&d_u, size) != hipSuccess) {
        printf("Erro ao alocar memória para d_u: %s\n", hipGetErrorString(hipGetLastError()));
    }
    if (hipMallocManaged((void**)&d_v, size) != hipSuccess) {
        printf("Erro ao alocar memória para d_v: %s\n", hipGetErrorString(hipGetLastError()));
    }
    if (hipMallocManaged((void**)&d_w, size) != hipSuccess) {
        printf("Erro ao alocar memória para d_w: %s\n", hipGetErrorString(hipGetLastError()));
    }
    if (hipMallocManaged((void**)&d_u_prev, size) != hipSuccess) {
        printf("Erro ao alocar memória para d_u_prev: %s\n", hipGetErrorString(hipGetLastError()));
    }
    if (hipMallocManaged((void**)&d_v_prev, size) != hipSuccess) {
        printf("Erro ao alocar memória para d_v_prev: %s\n", hipGetErrorString(hipGetLastError()));
    }
    if (hipMallocManaged((void**)&d_w_prev, size) != hipSuccess) {
        printf("Erro ao alocar memória para d_w_prev: %s\n", hipGetErrorString(hipGetLastError()));
    }
    if (hipMallocManaged((void**)&d_dens, size) != hipSuccess) {
        printf("Erro ao alocar memória para d_dens: %s\n", hipGetErrorString(hipGetLastError()));
    }
    if (hipMallocManaged((void**)&d_dens_prev, size) != hipSuccess) {
        printf("Erro ao alocar memória para d_dens_prev: %s\n", hipGetErrorString(hipGetLastError()));
    }
}

void cudaHostToDevice(float* u, float* v, float* w, float* u_prev, float* v_prev, float* w_prev, float* dens, float* dens_prev){
    int size = global_size * sizeof(float);
    if(u != nullptr) hipMemcpy(d_u, u, size, hipMemcpyHostToDevice);
    if(v != nullptr) hipMemcpy(d_v, v, size, hipMemcpyHostToDevice);
    if(w != nullptr) hipMemcpy(d_w, w, size, hipMemcpyHostToDevice);
    if(u_prev != nullptr) hipMemcpy(d_u_prev, u_prev, size, hipMemcpyHostToDevice);
    if(v_prev != nullptr) hipMemcpy(d_v_prev, v_prev, size, hipMemcpyHostToDevice);
    if(w_prev != nullptr) hipMemcpy(d_w_prev, w_prev, size, hipMemcpyHostToDevice);

    if(dens != nullptr) hipMemcpy(d_dens, dens, size, hipMemcpyHostToDevice);
    if(dens_prev != nullptr) hipMemcpy(d_dens_prev, dens_prev, size, hipMemcpyHostToDevice);
}

void cudaDeviceToHost(float* u, float* v, float* w, float* u_prev, float* v_prev, float* w_prev, float* dens, float* dens_prev){
    int size = global_size * sizeof(float);
    if(u != nullptr) hipMemcpy(u, d_u, size, hipMemcpyDeviceToHost);
    if(v != nullptr) hipMemcpy(v, d_v, size, hipMemcpyDeviceToHost);
    if(w != nullptr) hipMemcpy(w, d_w, size, hipMemcpyDeviceToHost);
    if(u_prev != nullptr) hipMemcpy(u_prev, d_u_prev, size, hipMemcpyDeviceToHost);
    if(v_prev != nullptr) hipMemcpy(v_prev, d_v_prev, size, hipMemcpyDeviceToHost);
    if(w_prev != nullptr) hipMemcpy(w_prev, d_w_prev, size, hipMemcpyDeviceToHost);

    if(dens != nullptr) hipMemcpy(dens, d_dens, size, hipMemcpyDeviceToHost);
    if(dens_prev != nullptr) hipMemcpy(dens_prev, d_dens_prev, size, hipMemcpyDeviceToHost);
}

// Liberta os mallocs constantes
void freeCudaMalloc(){
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_u_prev);
    hipFree(d_v_prev);
    hipFree(d_w_prev);

    hipFree(d_dens);
    hipFree(d_dens_prev);
}

__global__ void add_source_kernel(int M, int N, int O, float *x, float *s, float dt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = (M + 2) * (N + 2) * (O + 2);

    if (idx < size) {
        x[idx] += dt * s[idx];
    }
}

// Para já mais rapida que o kernel então é a utilizada
void add_source(int M, int N, int O, float *x, float *s, float dt) {
    int threadsPerBlock = 256;
    int numBlocks = (global_size + threadsPerBlock - 1) / threadsPerBlock;

    add_source_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, x, s, dt);
    hipDeviceSynchronize();
    /*int size = (M + 2) * (N + 2) * (O + 2);
    for (int i = 0; i < size; i++) {
        x[i] += dt * s[i];
    }*/
}

__global__ void set_bnd_kernel(int M, int N, int O, int b, float* x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Bordas em k = 0 e k = O+1
    if (i >= 1 && i <= M && j >= 1 && j <= N) {
        if (k == 0) x[IX(i, j, 0)] = (b == 3) ? -x[IX(i, j, 1)] : x[IX(i, j, 1)];
        if (k == O + 1) x[IX(i, j, O + 1)] = (b == 3) ? -x[IX(i, j, O)] : x[IX(i, j, O)];
    }

    // Bordas em i = 0 e i = M+1
    if (j >= 1 && j <= N && k >= 1 && k <= O) {
        if (i == 0) x[IX(0, j, k)] = (b == 1) ? -x[IX(1, j, k)] : x[IX(1, j, k)];
        if (i == M + 1) x[IX(M + 1, j, k)] = (b == 1) ? -x[IX(M, j, k)] : x[IX(M, j, k)];
    }

    // Bordas em j = 0 e j = N+1
    if (i >= 1 && i <= M && k >= 1 && k <= O) {
        if (j == 0) x[IX(i, 0, k)] = (b == 2) ? -x[IX(i, 1, k)] : x[IX(i, 1, k)];
        if (j == N + 1) x[IX(i, N + 1, k)] = (b == 2) ? -x[IX(i, N, k)] : x[IX(i, N, k)];
    }

    // Cálculo explícito dos cantos
    if (i == 0 && j == 0 && k == 0) 
        x[IX(0, 0, 0)] = 0.33f * (x[IX(1, 0, 0)] + x[IX(0, 1, 0)] + x[IX(0, 0, 1)]);
    if (i == M + 1 && j == 0 && k == 0) 
        x[IX(M + 1, 0, 0)] = 0.33f * (x[IX(M, 0, 0)] + x[IX(M + 1, 1, 0)] + x[IX(M + 1, 0, 1)]);
    if (i == 0 && j == N + 1 && k == 0) 
        x[IX(0, N + 1, 0)] = 0.33f * (x[IX(1, N + 1, 0)] + x[IX(0, N, 0)] + x[IX(0, N + 1, 1)]);
    if (i == M + 1 && j == N + 1 && k == 0) 
        x[IX(M + 1, N + 1, 0)] = 0.33f * (x[IX(M, N + 1, 0)] + x[IX(M + 1, N, 0)] + x[IX(M + 1, N + 1, 1)]);
}

// Acho que não é utilizada neste momento. Todas as funções chamam diretamente o kernel
void set_bnd(int M, int N, int O, int b, float *x) {
    // Configuração dos kernels
    int size = global_size * sizeof(float);
    dim3 threadsPerBlock(64, 8, 2);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (O + threadsPerBlock.z - 1) / threadsPerBlock.z);
    
    //hipMemcpy(new_x, x, size, hipMemcpyHostToDevice);

    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, x);
    hipDeviceSynchronize();

    //hipMemcpy(x, new_x, size, hipMemcpyDeviceToHost);

    /*int i, j;

    for (j = 1; j <= N; j++) {
        for (i = 1; i <= M; i++) {
            x[IX(i, j, 0)] = (b == 3) ? -x[IX(i, j, 1)] : x[IX(i, j, 1)];
            x[IX(i, j, O + 1)] = (b == 3) ? -x[IX(i, j, O)] : x[IX(i, j, O)];
        }
    }

    for (j = 1; j <= O; j++) {
        for (i = 1; i <= N; i++) {
            x[IX(0, i, j)] = (b == 1) ? -x[IX(1, i, j)] : x[IX(1, i, j)];
            x[IX(M + 1, i, j)] = (b == 1) ? -x[IX(M, i, j)] : x[IX(M, i, j)];
        }
    }

    for (j = 1; j <= O; j++) {
        for (i = 1; i <= M; i++) {
            x[IX(i, 0, j)] = (b == 2) ? -x[IX(i, 1, j)] : x[IX(i, 1, j)];
            x[IX(i, N + 1, j)] = (b == 2) ? -x[IX(i, N, j)] : x[IX(i, N, j)];
        }
    }

    x[IX(0, 0, 0)] = 0.33f * (x[IX(1, 0, 0)] + x[IX(0, 1, 0)] + x[IX(0, 0, 1)]); 
    x[IX(M + 1, 0, 0)] = 0.33f * (x[IX(M, 0, 0)] + x[IX(M + 1, 1, 0)] + x[IX(M + 1, 0, 1)]); 
    x[IX(0, N + 1, 0)] = 0.33f * (x[IX(1, N + 1, 0)] + x[IX(0, N, 0)] + x[IX(0, N + 1, 1)]); 
    x[IX(M + 1, N + 1, 0)] = 0.33f * (x[IX(M, N + 1, 0)] + x[IX(M + 1, N, 0)] + x[IX(M + 1, N + 1, 1)]); */
}

__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

__global__ void lin_solve_red_kernel(int M, int N, int O, int b, float* x, const float* x0, float a, float inv_c, float* max_change) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O) {
        if ((i + j + k) % 2 == 1) {  // Red
            int index = IX(i, j, k);
            float old_x = x[index];
            x[index] = (x0[index] +
                        a * (x[IX(i - 1, j, k)] + x[IX(i + 1, j, k)] +
                             x[IX(i, j - 1, k)] + x[IX(i, j + 1, k)] +
                             x[IX(i, j, k - 1)] + x[IX(i, j, k + 1)])) * inv_c;
            float change = fabsf(x[index] - old_x);
            //atomicMaxFloat(max_change, change);
            if (change > *max_change) *max_change = change;
        }
    }
}

__global__ void lin_solve_black_kernel(int M, int N, int O, int b, float* x, const float* x0, float a, float inv_c, float* max_change) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O) {
        if ((i + j + k) % 2 == 0) {  // Black
            int index = IX(i, j, k);
            float old_x = x[index];
            x[index] = (x0[index] +
                        a * (x[IX(i - 1, j, k)] + x[IX(i + 1, j, k)] +
                             x[IX(i, j - 1, k)] + x[IX(i, j + 1, k)] +
                             x[IX(i, j, k - 1)] + x[IX(i, j, k + 1)])) * inv_c;
            float change = fabsf(x[index] - old_x);
            //atomicMaxFloat(max_change, change);
            if (change > *max_change) *max_change = change;
        }
    }
}

void lin_solve(int M, int N, int O, int b, float* x, const float* x0, float a, float c) {
    float tol = 1e-7f;
    float max_change;
    float* d_max_change;

    hipMallocManaged((void**)&d_max_change, sizeof(float));

    // Configuração dos kernels
    dim3 threadsPerBlock(64, 8, 2);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (O + threadsPerBlock.z - 1) / threadsPerBlock.z);

    float inv_c = 1.0f / c;
    int iterations = 0;

    // Iterar até atingir a tolerância
    do {
        *d_max_change = 0.0f;

        // Fase Red
        lin_solve_red_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, x, x0, a, inv_c, d_max_change);
        hipDeviceSynchronize();

        // Fase Black
        lin_solve_black_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, x, x0, a, inv_c, d_max_change);
        hipDeviceSynchronize();

        // Aplicar condições de contorno (não é preciso chamar o setup porque o array já está na GPU)
        set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, x);
        hipDeviceSynchronize();

    } while (*d_max_change > tol && ++iterations < 20);
}


void diffuse(int M, int N, int O, int b, float *x, float *x0, float diff, float dt) {
    int max = MAX(M, MAX(N, O));
    float a = dt * diff * max * max;
    lin_solve(M, N, O, b, x, x0, a, 1 + 6 * a);
}

__global__ void advect_kernel(int M, int N, int O, int b, float* d, const float* d0, const float* u, const float* v, const float* w, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1; // +1 para evitar bordas
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i > M || j > N || k > O) return;

    int index = IX(i, j, k);
    float dtX = dt * M, dtY = dt * N, dtZ = dt * O;

    float u_val = u[index], v_val = v[index], w_val = w[index];
    float x = i - dtX * u_val, y = j - dtY * v_val, z = k - dtZ * w_val;

    x = (x < 0.5f) ? 0.5f : (x > M + 0.5f) ? M + 0.5f : x;
    y = (y < 0.5f) ? 0.5f : (y > N + 0.5f) ? N + 0.5f : y;
    z = (z < 0.5f) ? 0.5f : (z > O + 0.5f) ? O + 0.5f : z;

    int i0 = (int)x, i1 = i0 + 1, j0 = (int)y, j1 = j0 + 1, k0 = (int)z, k1 = k0 + 1;
    float s1 = x - i0, s0 = 1 - s1, t1 = y - j0, t0 = 1 - t1, u1 = z - k0, u0 = 1 - u1;

    d[index] = 
        s0 * (t0 * (u0 * d0[IX(i0, j0, k0)] + u1 * d0[IX(i0, j0, k1)]) + 
              t1 * (u0 * d0[IX(i0, j1, k0)] + u1 * d0[IX(i0, j1, k1)])) +
        s1 * (t0 * (u0 * d0[IX(i1, j0, k0)] + u1 * d0[IX(i1, j0, k1)]) + 
              t1 * (u0 * d0[IX(i1, j1, k0)] + u1 * d0[IX(i1, j1, k1)]));
}

void advect(int M, int N, int O, int b, float* h_d, float* h_d0, float* h_u, float* h_v, float* h_w, float dt) {
    // Configuração de dimensões dos blocos e grades
    dim3 threadsPerBlock(64, 8, 2);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (O + threadsPerBlock.z - 1) / threadsPerBlock.z);

    // Lançar o kernel
    advect_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, h_d, h_d0, h_u, h_v, h_w, dt);
    hipDeviceSynchronize();

    // Aplicar condições de contorno (não é preciso chamar o setup porque o array já está na GPU)
    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, h_d);
    hipDeviceSynchronize();
}

__global__ void project_divergence_kernel(int M, int N, int O, float* u, float* v, float* w, float* p, float* div, float inv_max_dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O) {
        int index = IX(i, j, k);
        div[index] = -0.5f * (
            u[IX(i + 1, j, k)] - u[IX(i - 1, j, k)] +
            v[IX(i, j + 1, k)] - v[IX(i, j - 1, k)] +
            w[IX(i, j, k + 1)] - w[IX(i, j, k - 1)]
        ) * inv_max_dim;

        p[index] = 0.0f;
    }
}

__global__ void project_update_velocity_kernel(int M, int N, int O, float* u, float* v, float* w, float* p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O) {
        int index = IX(i, j, k);
        u[index] -= 0.5f * (p[IX(i + 1, j, k)] - p[IX(i - 1, j, k)]);
        v[index] -= 0.5f * (p[IX(i, j + 1, k)] - p[IX(i, j - 1, k)]);
        w[index] -= 0.5f * (p[IX(i, j, k + 1)] - p[IX(i, j, k - 1)]);
    }
}

// Projection step to ensure incompressibility (make the velocity field
// divergence-free)
void project(int M, int N, int O, float* h_u, float* h_v, float* h_w, float* h_p, float* h_div) {
    // Configuração de dimensões dos blocos e grades
    dim3 threadsPerBlock(64, 8, 2);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (O + threadsPerBlock.z - 1) / threadsPerBlock.z);

    float inv_max_dim = 1.0f / max(M, max(N, O));

    // Calcular divergência e inicializar pressão
    project_divergence_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, h_u, h_v, h_w, h_p, h_div, inv_max_dim);
    hipDeviceSynchronize();

    // Aplicar condições de contorno (não é preciso chamar o setup porque o array já está na GPU)
    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 0, h_div);
    hipDeviceSynchronize();

    // Aplicar condições de contorno (não é preciso chamar o setup porque o array já está na GPU)
    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 0, h_p);
    hipDeviceSynchronize();

    // Resolver equação linear para pressão
    lin_solve(M, N, O, 0, h_p, h_div, 1, 6);

    // Atualizar campos de velocidade
    project_update_velocity_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, h_u, h_v, h_w, h_p);
    hipDeviceSynchronize();

    // Ajustar bordas para os campos de velocidade
    // Aplicar condições de contorno (não é preciso chamar o setup porque o array já está na GPU)
    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 1, h_u);
    hipDeviceSynchronize();

    // Aplicar condições de contorno (não é preciso chamar o setup porque o array já está na GPU)
    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 2, h_v);
    hipDeviceSynchronize();

    // Aplicar condições de contorno (não é preciso chamar o setup porque o array já está na GPU)
    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 3, h_w);
    hipDeviceSynchronize();
}

// Step function for density
void dens_step(int M, int N, int O, float *x, float *x0, float *u, float *v,
               float *w, float diff, float dt) {
  add_source(M, N, O, d_dens, d_dens_prev, dt);
  SWAP(d_dens_prev, d_dens);
  diffuse(M, N, O, 0, d_dens, d_dens_prev, diff, dt);
  SWAP(d_dens_prev, d_dens);
  advect(M, N, O, 0, d_dens, d_dens_prev, d_u, d_v, d_w, dt);
}

// Step function for velocity
void vel_step(int M, int N, int O, float *u, float *v, float *w, float *u0,
              float *v0, float *w0, float visc, float dt) {
  add_source(M, N, O, d_u, d_u_prev, dt);
  add_source(M, N, O, d_v, d_v_prev, dt);
  add_source(M, N, O, d_w, d_w_prev, dt);
  SWAP(d_u_prev, d_u);
  diffuse(M, N, O, 1, d_u, d_u_prev, visc, dt);
  SWAP(d_v_prev, d_v);
  diffuse(M, N, O, 2, d_v, d_v_prev, visc, dt);
  SWAP(d_w_prev, d_w);
  diffuse(M, N, O, 3, d_w, d_w_prev, visc, dt);
  project(M, N, O, d_u, d_v, d_w, d_u_prev, d_v_prev);
  SWAP(d_u_prev, d_u);
  SWAP(d_v_prev, d_v);
  SWAP(d_w_prev, d_w);
  advect(M, N, O, 1, d_u, d_u_prev, d_u_prev, d_v_prev, d_w_prev, dt);
  advect(M, N, O, 2, d_v, d_v_prev, d_u_prev, d_v_prev, d_w_prev, dt);
  advect(M, N, O, 3, d_w, d_w_prev, d_u_prev, d_v_prev, d_w_prev, dt);
  project(M, N, O, d_u, d_v, d_w, d_u_prev, d_v_prev);
}